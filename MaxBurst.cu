#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
__global__   void burst(float *dx, int n, int k, float *dxbar, int maxWinSize) {
   int tid=threadIdx.y*blockDim.x+threadIdx.x;
   int me=blockIdx.x*blockDim.x*blockDim.y+tid;
   int width=n-k+1;
   int x=me%width;
   int y=me/width;
   int perstart=x;//start
   int perend;
   int indx=0;
   //extern __shared__ float sx[];
   int perlen=y+k;//length of window, or window size. Notice if minimum windowSize k is smaller than n/2 ,we only need maximum windowSize to be 2k.
   //each thread copy one number to shared memory, notice we have more threads than numbers/
   indx=perstart*(n-k+1)+perlen-k;
   dxbar[indx]=-1000.0;
   /*
   if(me<n){
      sx[me]=dx[me];
   }
   __syncthreads();
   */
   if(maxWinSize>n-perstart){
     maxWinSize=n-perstart;
   } 
   if (perstart<=n-k && perlen>=k && perlen<=maxWinSize){
      perend=perstart+perlen-1;
      int i; float tot=0;
      for(i=perstart;i<=perend;i++) tot+=dx[i];
      dxbar[indx]=tot/(perend-perstart+1);
   }
   else{
      //printf("mean, indx=%f, %d\n", dxbar[indx], indx);
      return;
   }
   __syncthreads();
   //printf("mean,indx=%f, %d\n", dxbar[indx], indx);
}

__global__ void reduce(float *g_idata, float *g_odata){
   extern __shared__ float sdata[];
   int tid=threadIdx.y*blockDim.x+threadIdx.x;
   unsigned int i=blockIdx.x*blockDim.x*blockDim.y+tid;
 //  sdata[tid]=g_idata[i];
   //__syncthreads();
   //printf("sdata[tid],tid=%f, %d\n", sdata[tid], tid);
   for(unsigned int s=1; s<blockDim.x*blockDim.y; s*=2){
      if(tid%(2*s)==0){
         if(g_idata[i+s]>g_idata[i]){
            g_idata[i]=g_idata[i+s];
         }
      }
      __syncthreads();
   }

   if(tid==0) {g_odata[blockIdx.x]=g_idata[i];
   printf("in reduce, blockIdx.x, ans,%d %f,\n", blockIdx.x, g_odata[blockIdx.x]);
   }
}

// things need to fix probably: bigmax allocate one int; passing n and k and bigmax to cuda function
void maxburst(float *x, int n, int k, int *startend, float *bigmax){
    float *dx; //device x
    int asize = n*sizeof(float);
    float *out;//each block has an output max mean answer.
    float *dout; //on device, out.

    float* xbar; //Means for every possiblle start position, and window size.
    float* dxbar;
    int nblk=(n-k+1)*(n-k+1)/128+1;//Number of blocks
    int maxWinSize=n;
    // copy host matrix to device matrix

    xbar=(float *) malloc(sizeof(float)*(n-k+1)*(n-k+1));
    out=(float *) malloc(sizeof(float)*nblk);
    // allocate space for device matrix
    hipMalloc ((void **)&dx,asize);
    hipMalloc ((void **)&dxbar, sizeof(float)*(n-k+1)*(n-k+1));
    hipMalloc (( void **)&dout, nblk*sizeof(float));
    hipMemcpy(dx,x,asize ,hipMemcpyHostToDevice);
    hipMemcpy(dxbar,xbar,sizeof(float)*(n-k+1)*(n-k+1) ,hipMemcpyHostToDevice);
    hipMemcpy(dout, out, sizeof(float)*(nblk), hipMemcpyHostToDevice);

    // set up parameters for threads structure
    dim3 dimGrid(nblk,1); // n blocks
    dim3 dimBlock(8, 16,1);
    // invoke the ker
    // make winsize
    if(n>2*k){
       maxWinSize=2*k;
    }

    burst<<<dimGrid,dimBlock>>>(dx,n,k,dxbar, maxWinSize);
    hipDeviceSynchronize();
    //SomeReduce function
    reduce<<<dimGrid, dimBlock>>>(dxbar, dout);
    // copy row vector from device to host
    //hipMemcpy(xbar, dxbar, sizeof(float)*(n-k+1)*(n-k+1), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipMemcpy(out, dout, sizeof(float)*nblk, hipMemcpyDeviceToHost);

    for (int i=0; i<nblk; i++){
       //printf("%f\n,",out[i]);
       if (out[i]>bigmax[0]){
          bigmax[0]=out[i];
       }
    }
    printf("bigmax is%f\n", bigmax[0]);
    hipFree(dxbar);
    hipFree(dout);
    hipFree (dx);

}
int main(int arc, char **argv){
  float *x;
  int n=1000;
  int k=3;
  int *startend;
  float *bigmax;
  bigmax=(float*) malloc(sizeof(float));
  startend=(int*) malloc(sizeof(int)*2);
  x=(float*) malloc(sizeof(float)*n);
  int i;
  for(i=0; i<n; i++){
     x[i]=i*1.0;
  }
  bigmax[0]=0;
  maxburst(x, n, k, startend, bigmax);
} 
