#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
__global__   void SmallWindowBurst(float *dx, int n, int k, float *dxbar, float *dCandMeans, float *dPreCandMeans, float *sum, int *startm1){
   int tid=threadIdx.y*blockDim.x+threadIdx.x;
   int me =blockIdx.x*blockDim.x*blockDim.y+tid;//me [0 to n-k+1)
   int winsize=me+k;//[k to n+1)
   if(winsize>n){
      return;
   }
   //dxbar is of dimension, n-k+1
   if(n<2*k){
      return;
   }
   extern __shared__ float sx[];//of dimension, n
   //copy dx to sx
   sx[me]=dx[me];
   startm1[me]=0;
   if(me==0){
     for(int i=n-k+1; i<n; i++){
       sx[i]=dx[i];
     }
   }
   
   
   __syncthreads();

   dCandMeans[me]=sum[me]/winsize;
   dPreCandMeans[me]=sum[me]/winsize;
   //printf("av=%f, %d\n", dCandMeans[me], winsize);
   if(winsize==n){
      dxbar[winsize]=dCandMeans[me];
      return;
   }
   //now find rest of means, rolling window
   dxbar[me]=dCandMeans[me];
   for(; startm1[me]<(n-winsize); startm1[me]++){
      dPreCandMeans[me]=dCandMeans[me];
      dCandMeans[me]=dPreCandMeans[me]+((sx[winsize+startm1[me]]-sx[startm1[me]])/winsize);
      
     // printf("start, maxCand, n-winsize=%d, %f, %d\n", startm1[me]+1, dCandMeans[me], n-winsize );
      if(dCandMeans[me]>dxbar[me]){
         dxbar[me]=dCandMeans[me];
      }
   }
   //printf("%d\n",maxCand);
   
   printf("dxbar[winzie], winsize=%f, %d\n", dxbar[me], winsize);
}


__global__   void burst(float *dx, int n, int k, float *dxbar, int maxWinSize) {
   int tid=threadIdx.y*blockDim.x+threadIdx.x;
   int me=blockIdx.x*blockDim.x*blockDim.y+tid;
   int width=n-k+1;
   int x=me%width;
   int y=me/width;
   int perstart=x;//start
   int perend;
   int indx=0;
   extern __shared__ float sx[];
   int perlen=y+k;//length of window, or window size. Notice if minimum windowSize k is smaller than n/2 ,we only need maximum windowSize to be 2k.
   //each thread copy one number to shared memory, notice we have more threads than numbers/
   indx=perstart*(n-k+1)+perlen-k;
   dxbar[indx]=-1000.0;
   if(me<n){
      sx[me]=dx[me];
   }
   __syncthreads();
   if(maxWinSize>n-perstart){
     maxWinSize=n-perstart;
   } 
   if (perstart<=n-k && perlen>=k && perlen<=maxWinSize && n<2*k){
      perend=perstart+perlen-1;
      int i; float tot=0;
      for(i=perstart;i<=perend;i++) tot+=sx[i];
      dxbar[indx]=tot/(perend-perstart+1);
   }
   else{
      return;
   }
   __syncthreads();
   //printf("mean,indx=%f, %d\n", dxbar[indx], indx);
}

__global__ void reduce(float *g_idata, float *g_odata){
   extern __shared__ float sdata[];
   int tid=threadIdx.y*blockDim.x+threadIdx.x;
   unsigned int i=blockIdx.x*blockDim.x*blockDim.y+tid;
   sdata[tid]=g_idata[i];
   __syncthreads();
   //printf("sdata[tid],tid=%f, %d\n", sdata[tid], tid);
   for(unsigned int s=1; s<blockDim.x*blockDim.y; s*=2){
      int index=2*s*tid;
      if(index<blockDim.x*blockDim.y){
         if(sdata[index]<sdata[index+s]){
            sdata[index]=sdata[index+s];
         }
      }
      __syncthreads();

   }

   if(tid==0) {g_odata[blockIdx.x]=sdata[0];
   printf("in reduce, %f,\n", g_odata[blockIdx.x]);}
}

// things need to fix probably: bigmax allocate one int; passing n and k and bigmax to cuda function
void maxburst(float *x, int n, int k, int *startend, float *bigmax){
    float *dx; //device x
    float *dbigmax; //device bigmax
    int asize = n*sizeof(float);
    float *out;//each block has an output max mean answer.
    float *dout; //on device, out.

    float* xbar; //Means for every possiblle start position, and window size.
    float* dxbar;
    int nblk=(n-k+1)*(n-k+1)/256+1;//Number of blocks
    int maxWinSize=n;
    // copy host matrix to device matrix

    xbar=(float *) malloc(sizeof(float)*(n-k+1)*(n-k+1));
    out=(float *) malloc(sizeof(float)*nblk);
    // allocate space for device matrix
    hipMalloc ((void **)&dx,asize);
    hipMalloc (( void **)&dbigmax , sizeof(float) );
    hipMalloc ((void **)&dxbar, sizeof(float)*(n-k+1)*(n-k+1));
    hipMalloc (( void **)&dout, nblk*sizeof(float));
    hipMemcpy(dx,x,asize ,hipMemcpyHostToDevice);
    hipMemcpy(dbigmax, bigmax, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dout, out, sizeof(float)*(nblk), hipMemcpyHostToDevice);

    // invoke the ker
    // make winsize
    if(n<2*k){
       maxWinSize=2*k;
       nblk=(n-k+1)*(n-k+1)/256+1;
    }
    else{
       nblk=(n-k+1)/256+1;
    }
    dim3 dimGrid(nblk,1); // n blocks
    dim3 dimBlock(16, 16,1);
    if(n<2*k){
       hipMalloc ((void **)&dxbar, sizeof(float)*(n-k+1)*(n-k+1));
       hipMemcpy(dxbar,xbar,sizeof(float)*(n-k+1)*(n-k+1) ,hipMemcpyHostToDevice);
       burst<<<dimGrid,dimBlock, n*nblk>>>(dx,n,k,dxbar, maxWinSize);
    }
    else{
       nblk=(n-k+1)/256+1;
       hipMalloc ((void **)&dxbar, sizeof(float)*(n-k+1));
       hipMemcpy(dxbar,xbar,sizeof(float)*(n-k+1) ,hipMemcpyHostToDevice);
       
       float* CandMeans;
       CandMeans=(float*) malloc(sizeof(float)*(n-k+1));
       float* dCandMeans;
       hipMalloc ((void **)&dCandMeans, sizeof(float)*(n-k+1));
       hipMemcpy(dCandMeans, CandMeans, sizeof(float)*(n-k+1), hipMemcpyHostToDevice);
       
       float* PreCandMeans;
       PreCandMeans=(float*) malloc(sizeof(float)*(n-k+1));
       float* dPreCandMeans;
       hipMalloc ((void **)&dPreCandMeans, sizeof(float)*(n-k+1));
       hipMemcpy(dPreCandMeans, PreCandMeans, sizeof(float)*(n-k+1), hipMemcpyHostToDevice);
       
       float* Sums;
       Sums=(float*) malloc(sizeof(float)*(n-k+1));
       for(int i=0; i<k;i++){
          Sums[0]+=x[i];
       }
       for(int i=1; i<n-k+1;i++){
          Sums[i]=Sums[i-1]+x[i+k-1];
       }
       float* dSums;
       hipMalloc ((void **)&dSums, sizeof(float)*(n-k+1));
       hipMemcpy(dSums, Sums, sizeof(float)*(n-k+1), hipMemcpyHostToDevice);

       int* Startm1;
       int* dStartm1;
       Startm1=(int*) malloc(sizeof(int)*(n-k+1));
       hipMalloc ((void **)&dStartm1, sizeof(int)*(n-k+1));
       hipMemcpy(dStartm1, Startm1, sizeof(int)*(n-k+1), hipMemcpyHostToDevice);
       
       
       SmallWindowBurst<<<dimGrid, dimBlock, nblk*n>>>(dx, n, k, dxbar, dCandMeans, dPreCandMeans, dSums, dStartm1);
       hipFree(dSums);
       hipFree(dPreCandMeans);
       hipFree(dCandMeans);
       hipFree(dStartm1);
    }
    //If the wind size is smaller than n/2, we are goint to use first approach. n-k+1, in second senario, we have (n-k+1) **2
    
    
    hipDeviceSynchronize();
    hipMemcpy(xbar, dxbar, sizeof(float)*(n-k+1)*(n-k+1), hipMemcpyDeviceToHost);
    int tmp=0;
    for(tmp=0; tmp<(n-k+1)*(n-k+1); tmp++){
       //printf("after copy from GPU to CPU, mean, indx  are %f, %d\n", xbar[tmp], tmp);
    }
    hipMemcpy(dxbar,xbar,sizeof(float)*(n-k+1)*(n-k+1) ,hipMemcpyHostToDevice);
    //SomeReduce function
    reduce<<<dimGrid, dimBlock, (n-k+1)*(n-k+1)>>>(dxbar, dout);
    // copy row vector from device to host
    //hipMemcpy(xbar, dxbar, sizeof(float)*(n-k+1)*(n-k+1), hipMemcpyDeviceToHost);
    //hipMemcpy(bigmax,dbigmax, sizeof(float),hipMemcpyDeviceToHost);
    //hipMemcpy(out, dout, sizeof(float)*nblk, hipMemcpyDeviceToHost);
    //printf("bigmax is%f\n", xbar[0]);
    hipFree(dxbar);
    hipFree(dout);
    hipFree (dbigmax);
    hipFree (dx);

}
int main(int arc, char **argv){
  float *x;
  int n=100;
  int k=3;
  int *startend;
  float *bigmax;
  bigmax=(float*) malloc(sizeof(float));
  startend=(int*) malloc(sizeof(int)*2);
  x=(float*) malloc(sizeof(float)*n);
  int i;
  for(i=0; i<n; i++){
     x[i]=i*1.0;
  }
  bigmax[0]=0;
  maxburst(x, n, k, startend, bigmax);
} 
