#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
__global__   void SmallWindowBurst(float *dx, int n, int k, float *dxbar){
   int tid=threadIdx.y*blockDim.x+threadIdx.x;
   int me =blockIdx.x*blockDim.x*blockDim.y+tid;//me [0 to n-k+1)
   int winsize=me+k;//[k to n+1)
   float sum=0.0;
   float dCandMean=0.0;
   float dPreCandMean=0.0;
   if(winsize>n){
      return;
   }
   //dxbar is of dimension, n-k+1
   if(n<2*k){
      return;
   }
   extern __shared__ float sx[];//of dimension, n
   //copy dx to sx
   sx[me]=dx[me];
   int startm1=0;
   if(me==0){
     for(int i=n-k+1; i<n; i++){
       sx[i]=dx[i];
     }
   }
   __syncthreads();

   for(int i=0;i<winsize;i++){
      sum+=sx[i];
   }
   dxbar[me]=sum/winsize;
   //printf("av=%f, %d\n", dxbar[me], winsize);

   dCandMean=sum/winsize;
   dPreCandMean=sum/winsize;
   if(winsize==n){
      dxbar[me]=dCandMean;
      return;
   }
   //now find rest of means, rolling window
   dxbar[me]=dCandMean;
   for(; startm1<(n-winsize); startm1++){
      dPreCandMean=dCandMean;
      dCandMean=dPreCandMean+((sx[winsize+startm1]-sx[startm1])/winsize);
      if(winsize==3){ 
         printf("start, maxCand, n-winsize=%d, %f, %d\n", startm1+1, dCandMean, n-winsize );
      }
      if(dCandMean>dxbar[me]){
         dxbar[me]=dCandMean;
      }
   }
   //printf("%d\n",maxCand);
   
   //printf("dxbar[winzie], winsize=%f, %d\n", dxbar[me], winsize);
}


__global__   void burst(float *dx, int n, int k, float *dxbar, int maxWinSize) {
   int tid=threadIdx.y*blockDim.x+threadIdx.x;
   int me=blockIdx.x*blockDim.x*blockDim.y+tid;
   int width=n-k+1;
   int x=me%width;
   int y=me/width;
   int perstart=x;//start
   int perend;
   int indx=0;
   extern __shared__ float sx[];
   int perlen=y+k;//length of window, or window size. Notice if minimum windowSize k is smaller than n/2 ,we only need maximum windowSize to be 2k.
   //each thread copy one number to shared memory, notice we have more threads than numbers/
   indx=perstart*(n-k+1)+perlen-k;
   dxbar[indx]=-1000.0;
   if(me<n){
      sx[me]=dx[me];
   }
   __syncthreads();
   if(maxWinSize>n-perstart){
     maxWinSize=n-perstart;
   } 
   if (perstart<=n-k && perlen>=k && perlen<=maxWinSize && n<2*k){
      perend=perstart+perlen-1;
      int i; float tot=0;
      for(i=perstart;i<=perend;i++) tot+=sx[i];
      dxbar[indx]=tot/(perend-perstart+1);
   }
   else{
      return;
   }
   __syncthreads();
   //printf("mean,indx=%f, %d\n", dxbar[indx], indx);
}

__global__ void reduce(float *g_idata, float *g_odata){
   extern __shared__ float sdata[];
   int tid=threadIdx.y*blockDim.x+threadIdx.x;
   unsigned int i=blockIdx.x*blockDim.x*blockDim.y+tid;
   sdata[tid]=g_idata[i];
   __syncthreads();
   //printf("sdata[tid],tid=%f, %d\n", sdata[tid], tid);
   for(unsigned int s=1; s<blockDim.x*blockDim.y; s*=2){
      int index=2*s*tid;
      if(index<blockDim.x*blockDim.y){
         if(sdata[index]<sdata[index+s]){
            sdata[index]=sdata[index+s];
         }
      }
      __syncthreads();

   }

   if(tid==0) {g_odata[blockIdx.x]=sdata[0];
   printf("in reduce, %f,\n", g_odata[blockIdx.x]);}
}

// things need to fix probably: bigmax allocate one int; passing n and k and bigmax to cuda function
void maxburst(float *x, int n, int k, int *startend, float *bigmax){
    float *dx; //device x
    float *dbigmax; //device bigmax
    int asize = n*sizeof(float);
    float *out;//each block has an output max mean answer.
    float *dout; //on device, out.

    float* xbar; //Means for every possiblle start position, and window size.
    float* dxbar;
    int nblk=(n-k+1)*(n-k+1)/256+1;//Number of blocks
    int maxWinSize=n;
    // copy host matrix to device matrix

    xbar=(float *) malloc(sizeof(float)*(n-k+1)*(n-k+1));
    out=(float *) malloc(sizeof(float)*nblk);
    // allocate space for device matrix
    hipMalloc ((void **)&dx,asize);
    hipMalloc (( void **)&dbigmax , sizeof(float) );
    hipMalloc (( void **)&dout, nblk*sizeof(float));
    hipMemcpy(dbigmax, bigmax, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dout, out, sizeof(float)*(nblk), hipMemcpyHostToDevice);
    hipMemcpy(dx,x,sizeof(float)*n, hipMemcpyHostToDevice);

    // invoke the ker
    // make winsize
    if(n<2*k){
       maxWinSize=2*k;
       nblk=(n-k+1)*(n-k+1)/256+1;
    }
    else{
       nblk=(n-k+1)/256+1;
    }
    dim3 dimGrid(nblk,1); // n blocks
    dim3 dimBlock(16, 16,1);
    if(n<2*k){
       hipMalloc ((void **)&dxbar, sizeof(float)*(n-k+1)*(n-k+1));
       hipMemcpy(dxbar,xbar,sizeof(float)*(n-k+1)*(n-k+1) ,hipMemcpyHostToDevice);
       burst<<<dimGrid,dimBlock, n*nblk>>>(dx,n,k,dxbar, maxWinSize);
    }
    else{
       hipMalloc ((void **)&dxbar, sizeof(float)*(n-k+1));
       hipMemcpy(dxbar,xbar,sizeof(float)*(n-k+1) ,hipMemcpyHostToDevice);
       
       
       SmallWindowBurst<<<dimGrid, dimBlock, nblk*n>>>(dx, n, k, dxbar);
    }
    //If the wind size is smaller than n/2, we are goint to use first approach. n-k+1, in second senario, we have (n-k+1) **2
    
    
    hipDeviceSynchronize();
    hipMemcpy(xbar, dxbar, sizeof(float)*(n-k+1)*(n-k+1), hipMemcpyDeviceToHost);
    int tmp=0;
    for(tmp=0; tmp<(n-k+1)*(n-k+1); tmp++){
       //printf("after copy from GPU to CPU, mean, indx  are %f, %d\n", xbar[tmp], tmp);
    }
    hipMemcpy(dxbar,xbar,sizeof(float)*(n-k+1)*(n-k+1) ,hipMemcpyHostToDevice);
    //SomeReduce function
    reduce<<<dimGrid, dimBlock, (n-k+1)*(n-k+1)>>>(dxbar, dout);
    // copy row vector from device to host
    //hipMemcpy(xbar, dxbar, sizeof(float)*(n-k+1)*(n-k+1), hipMemcpyDeviceToHost);
    //hipMemcpy(bigmax,dbigmax, sizeof(float),hipMemcpyDeviceToHost);
    //hipMemcpy(out, dout, sizeof(float)*nblk, hipMemcpyDeviceToHost);
    //printf("bigmax is%f\n", xbar[0]);
    hipFree(dxbar);
    hipFree(dout);
    hipFree (dbigmax);
    hipFree (dx);

}
int main(int arc, char **argv){
  float *x;
  int n=80;
  int k=3;
  int *startend;
  float *bigmax;
  bigmax=(float*) malloc(sizeof(float));
  startend=(int*) malloc(sizeof(int)*2);
  x=(float*) malloc(sizeof(float)*n);
  int i;
  for(i=0; i<n; i++){
     x[i]=i*1.0;
  }
  bigmax[0]=0;
  maxburst(x, n, k, startend, bigmax);
} 
